#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>

#include <thrust/sort.h>


#define OP_UNION 1
#define OP_INTERSECTION 2
#define OP_DIFFERENCE 0
#define OP_SPHERE 3

#define GET_OP_ID(flag) (flag & 0b11)
#define SET_OP_ID(flag, op) (flag = (flag & 0xfffffff8) | op)

#define IS_NEW_SEGMENT(i) (scene.tree.flags[i] & 0b100)

#define GET_INTERSECTION_ID_FROM_FLAG(flag) ((flag >> 16) | ((threadIdx.x / scene.tree.n) << 16))
#define GET_INTERSECTION_ID(i) GET_INTERSECTION_ID_FROM_FLAG(scene.tree.flags[i])
#define GET_INTERSECTION_COUNT(i) ((scene.tree.flags[i] & 0xffff) >> 3) 

#define MAKE_FLAG(intId, intCount, newSeg, op) ((intId << 16) | (intCount << 3) | (newSeg << 2) | op)

//#define FRAGMENT_GREATER(val1, i1, val2, i2) (GET_INTERSECTION_ID(i1) > GET_INTERSECTION_ID(i2)) || (GET_INTERSECTION_ID(i1) == GET_INTERSECTION_ID(i2) && val1 > val2)
#define FRAGMENT_GREATER(val1, i1, val2, i2) (i1 > i2) || (i1 == i2 && val1 > val2)

struct normalisedCsgTree
{
    int n;
    int* flags;
    int* color;
    float* x, * y, * z;
    float* radius;
};

#define TREE_SIZE(t) (2 * sizeof(int) + 4 * sizeof(float)) * t.n
#define TREE_INTS(t) 2 * t.n
#define TREE_FLOATS(t) 4 * t.n

struct light
{
    float ka, kd, ks, m;
};

struct sceneParameters
{
    normalisedCsgTree tree;
    float3 observer;
    float3 direction;
    float3 planeHorizontal;
    float3 unitVectorToLight;
    int width;
    int height;
    float planeHeight;
    float planeDistance;
    light light;
};


__constant__ struct sceneParameters scene {};

//const __device__ float planeHeight = 1, planeDistance = 0.5; // planeDistance - odległość od obserwatora do rzutni

__device__ float3 normalised(float3 &vec)
{
    float d = sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
    return make_float3(vec.x / d, vec.y / d, vec.z / d);
}

__device__ float3 minusNormalised(float3 &vec)
{
    float d = -sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
    return make_float3(vec.x / d, vec.y / d, vec.z / d);
}

__device__ float length(float3 &vec)
{
    return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float scalarProduct(float3 &v1, float3 &v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ void calculateIntersection(int i, float *frontFace, float *backFace, float3 &vecTopointOnScene)
{
    float x = scene.tree.x[i], y = scene.tree.y[i], z = scene.tree.z[i], radius = scene.tree.radius[i];
    //http://paulbourke.net/geometry/circlesphere/index.html#linesphere
    float a = vecTopointOnScene.x * vecTopointOnScene.x + vecTopointOnScene.y * vecTopointOnScene.y + vecTopointOnScene.z * vecTopointOnScene.z,
        b = 2 * (vecTopointOnScene.x * (scene.observer.x - x) + vecTopointOnScene.y * (scene.observer.y - y) + vecTopointOnScene.z * (scene.observer.z - z)),
        c = x * x + y * y + z * z + scene.observer.x * scene.observer.x + scene.observer.y * scene.observer.y + scene.observer.z * scene.observer.z - 2 * (x * scene.observer.x + y * scene.observer.y + z * scene.observer.z) - radius * radius;

    float delta = b * b - 4 * a * c;
    if (delta >= 0)
    {
        *frontFace = (-b - sqrt(delta)) / 2 / a;
        *backFace = (-b + sqrt(delta)) / 2 / a;
    }
    else
    {
        *frontFace = INFINITY;
        *backFace = -INFINITY;
    }
}


template <class T>
__device__ void swap(T &x, T &y)
{
    T p = x;
    x = y;
    y = p;
}

// fragment-sorts shm_faces of one block size
__device__ void fragmentSortInBlock(volatile float* shm_faces, volatile int* shm_indexes, volatile int* shm_intersectionIds, volatile int* shm_a_to_scan)
{
    const unsigned int tid = threadIdx.x;
    __syncthreads();
    for (unsigned int k = 2; k <= blockDim.x; k <<= 1)
        for (unsigned int j = k >> 1; j > 0; j >>= 1)
        {
            unsigned int ixj = tid ^ j;
            if (ixj > tid) {
                if ((tid & k) == 0) {
                    //if (FRAGMENT_GREATER(shm_faces[shm_perm[tid]], shm_intersectionIds[shm_perm[tid]], shm_faces[shm_perm[ixj]], shm_intersectionIds[shm_perm[ixj]]))
                    if (FRAGMENT_GREATER(shm_faces[tid], shm_intersectionIds[tid], shm_faces[ixj], shm_intersectionIds[ixj]))
                    {
                        //swap(shm_perm[tid], shm_perm[ixj]);
                        swap(shm_faces[tid], shm_faces[ixj]);
                        swap(shm_indexes[tid], shm_indexes[ixj]);
                        swap(shm_intersectionIds[tid], shm_intersectionIds[ixj]);
                        swap(shm_a_to_scan[tid], shm_a_to_scan[ixj]);
                    }
                }
                else {
                    //if (FRAGMENT_GREATER(shm_faces[shm_perm[ixj]], shm_intersectionIds[shm_perm[ixj]], shm_faces[shm_perm[tid]], shm_intersectionIds[shm_perm[tid]]))
                    if (FRAGMENT_GREATER(shm_faces[ixj], shm_intersectionIds[ixj], shm_faces[tid], shm_intersectionIds[tid]))
                    {
                        //swap(shm_perm[tid], shm_perm[ixj]);
                        swap(shm_faces[tid], shm_faces[ixj]);
                        swap(shm_indexes[tid], shm_indexes[ixj]);
                        swap(shm_intersectionIds[tid], shm_intersectionIds[ixj]); 
                        swap(shm_a_to_scan[tid], shm_a_to_scan[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    __syncthreads();
}
//__device__ void fragmentSortInBlockC(volatile float* shm_faces, volatile int* shm_indexes, volatile int* shm_intersectionIds, volatile int* shm_a_to_scan)
//{
//    const unsigned int tid = threadIdx.x;
//    __syncthreads();
//    for (unsigned int k = 2; k <= blockDim.x; k <<= 1)
//        for (unsigned int j = k >> 1; j > 0; j >>= 1)
//        {
//            unsigned int ixj = tid ^ j;
//            printf("%d %d %d %d\n", shm_indexes[tid], shm_indexes[ixj], tid, ixj);
//            if (ixj > tid) {
//                if ((tid & k) == 0) {
//                    if (FRAGMENT_GREATER(shm_faces[tid], shm_intersectionIds[tid], shm_faces[ixj], shm_intersectionIds[ixj]))
//                    {
//                        swap(shm_faces[tid], shm_faces[ixj]);
//                        swap(shm_indexes[tid], shm_indexes[ixj]);
//                        swap(shm_intersectionIds[tid], shm_intersectionIds[ixj]);
//                        swap(shm_a_to_scan[tid], shm_a_to_scan[ixj]);
//                    }
//                }
//                else {
//                    if (FRAGMENT_GREATER(shm_faces[ixj], shm_intersectionIds[ixj], shm_faces[tid], shm_intersectionIds[tid]))
//                    {
//                        swap(shm_faces[tid], shm_faces[ixj]);
//                        swap(shm_indexes[tid], shm_indexes[ixj]);
//                        swap(shm_intersectionIds[tid], shm_intersectionIds[ixj]);
//                        swap(shm_a_to_scan[tid], shm_a_to_scan[ixj]);
//                    }
//                }
//            }
//            __syncthreads();
//        }
//    __syncthreads();
//}

// merges shm_faces of 2*blockSize, when shm_faces[0...blockSize-1] and shm_faces[blockSize,2*blockSize-1] are fragment-sorted
__device__ void mergeTwoBlockSizeArrays(volatile float* shm_faces, volatile int* shm_indexes, volatile int* shm_intersectionIds, volatile int* shm_a_to_scan)
{
    const unsigned int tid = threadIdx.x;
    __syncthreads();
    //if (FRAGMENT_GREATER(shm_faces[shm_perm[tid]], shm_intersectionIds[shm_perm[tid]], shm_faces[shm_perm[2 * blockDim.x - tid - 1]], shm_intersectionIds[shm_perm[2 * blockDim.x - tid - 1]]))
    if (FRAGMENT_GREATER(shm_faces[tid], shm_intersectionIds[tid], shm_faces[2 * blockDim.x - tid - 1], shm_intersectionIds[2 * blockDim.x - tid - 1]))
    {
        
        //swap(shm_perm[tid], shm_perm[2 * blockDim.x - tid - 1]);
        swap(shm_faces[tid], shm_faces[2 * blockDim.x - tid - 1]);
        swap(shm_indexes[tid], shm_indexes[2 * blockDim.x - tid - 1]);
        swap(shm_intersectionIds[tid], shm_intersectionIds[2 * blockDim.x - tid - 1]);
        swap(shm_a_to_scan[tid], shm_a_to_scan[2 * blockDim.x - tid - 1]);
    }
    __syncthreads();
    for (unsigned int j = blockDim.x >> 1; j > 0; j >>= 1)
    {
        unsigned int ixj = tid ^ j;
        if (ixj > tid) {
            if (FRAGMENT_GREATER(shm_faces[tid], shm_intersectionIds[tid], shm_faces[ixj], shm_intersectionIds[ixj]))
            //if (FRAGMENT_GREATER(shm_faces[shm_perm[tid]], shm_intersectionIds[shm_perm[tid]], shm_faces[shm_perm[ixj]], shm_intersectionIds[shm_perm[ixj]]))
            {
                //swap(shm_perm[tid], shm_perm[ixj]);
                swap(shm_faces[tid], shm_faces[ixj]);
                swap(shm_indexes[tid], shm_indexes[ixj]);
                swap(shm_intersectionIds[tid], shm_intersectionIds[ixj]);
                swap(shm_a_to_scan[tid], shm_a_to_scan[ixj]);
            }
        }
        else {
            ixj += blockDim.x;
            //if (FRAGMENT_GREATER(shm_faces[shm_perm[ixj]], shm_intersectionIds[shm_perm[ixj]], shm_faces[shm_perm[tid + blockDim.x]], shm_intersectionIds[shm_perm[tid + blockDim.x]]))
            if (FRAGMENT_GREATER(shm_faces[ixj], shm_intersectionIds[ixj], shm_faces[tid + blockDim.x], shm_intersectionIds[tid + blockDim.x]))
            {
                //swap(shm_perm[ixj], shm_perm[tid + blockDim.x]);
                swap(shm_faces[tid + blockDim.x], shm_faces[ixj]);
                swap(shm_indexes[tid + blockDim.x], shm_indexes[ixj]);
                swap(shm_intersectionIds[tid + blockDim.x], shm_intersectionIds[ixj]);
                swap(shm_a_to_scan[tid + blockDim.x], shm_a_to_scan[ixj]);
            }
        }

        __syncthreads();
    }
}

// z pracy Efficient Parallel Sacn Algorithms for GPUs, S. Sengupta, M. Harris, M. Garland
__device__ int segscan_warp(volatile int* ptr, volatile int* hd)
{
    const unsigned int idx = threadIdx.x, lane = idx & 31;
    if (lane >= 1) {
        ptr[idx] = hd[idx] ? ptr[idx] : (ptr[idx - 1] + ptr[idx]);
        hd[idx] = hd[idx - 1] | hd[idx];
    }
    if (lane >= 2) {
        ptr[idx] = hd[idx] ? ptr[idx] : (ptr[idx - 2] + ptr[idx]);
        hd[idx] = hd[idx - 2] | hd[idx];
    }
    if (lane >= 4) {
        ptr[idx] = hd[idx] ? ptr[idx] : (ptr[idx - 4] + ptr[idx]);
        hd[idx] = hd[idx - 4] | hd[idx];
    }
    if (lane >= 8) {
        ptr[idx] = hd[idx] ? ptr[idx] : (ptr[idx - 8] + ptr[idx]);
        hd[idx] = hd[idx - 8] | hd[idx];
    }
    if (lane >= 16) {
        ptr[idx] = hd[idx] ? ptr[idx] : (ptr[idx - 16] + ptr[idx]);
        hd[idx] = hd[idx - 16] | hd[idx];
    }
    return ptr[idx];
}

__device__ int segscan_block(volatile int* ptr, volatile int* hd)
{
    const unsigned int idx = threadIdx.x;
    unsigned int warpid = idx >> 5;
    unsigned int warp_first = warpid << 5;
    unsigned int warp_last = warp_first + 31;
    // Step 1a:
    // Before overwriting the input head flags , record whether
    // this warp begins with an "open" segment.
    bool warp_is_open = (hd[warp_first] == 0);
    __syncthreads();
    // Step 1b:
    // Intra -warp segmented scan in each warp.
    int val = segscan_warp(ptr, hd);
    // Step 2a:
    // Since ptr [] contains *inclusive * results , irrespective of Kind ,
    // the last value is the correct partial result.
    int warp_total = ptr[warp_last];
    // Step 2b:
    // warp_flag is the OR -reduction of the flags in a warp and is
    // computed indirectly from the mindex values in hd [].
    // will_accumulate indicates that a thread will only accumulate a
    // partial result in Step 4 if there is no segment boundary to its left.
    bool warp_flag = hd[warp_last] != 0 || !warp_is_open;
    bool will_accumulate = warp_is_open && hd[idx] == 0;
    __syncthreads();
    // Step 2c: The last thread in each warp writes partial results
    if (idx == warp_last)
    {
        ptr[warpid] = warp_total;
        hd[warpid] = warp_flag;
    }
    __syncthreads();
    // Step 3: One warp scans the per -warp results
    if (warpid == 0)
        segscan_warp(ptr, hd);
    __syncthreads();
    // Step 4: Accumulate results from Step 3, as appropriate .
    if (warpid != 0 && will_accumulate)
        val = ptr[warpid - 1] + val;
    __syncthreads();
    ptr[idx] = val;
    __syncthreads();
    return val;
}

__device__ float segscan_min_warp(volatile float* ptr, volatile int* hd, volatile int* indexes)
{
    const unsigned int idx = threadIdx.x, lane = idx & 31;
    /*if (lane >= 1) {
        ptr[idx] = hd[idx] ? ptr[idx] : fminf(ptr[idx - 1], ptr[idx]);
        hd[idx] = hd[idx - 1] | hd[idx];
    }
    if (lane >= 2) {
        ptr[idx] = hd[idx] ? ptr[idx] : fminf(ptr[idx - 2], ptr[idx]);
        hd[idx] = hd[idx - 2] | hd[idx];
    }
    if (lane >= 4) {
        ptr[idx] = hd[idx] ? ptr[idx] : fminf(ptr[idx - 4], ptr[idx]);
        hd[idx] = hd[idx - 4] | hd[idx];
    }
    if (lane >= 8) {
        ptr[idx] = hd[idx] ? ptr[idx] : fminf(ptr[idx - 8], ptr[idx]);
        hd[idx] = hd[idx - 8] | hd[idx];
    }
    if (lane >= 16) {
        ptr[idx] = hd[idx] ? ptr[idx] : fminf(ptr[idx - 16], ptr[idx]);
        hd[idx] = hd[idx - 16] | hd[idx];
    }*/
    if (lane >= 1)
    {
        if (!hd[idx] && ptr[idx - 1] < ptr[idx])
        {
            ptr[idx] = ptr[idx - 1];
            indexes[idx] = indexes[idx - 1];
        }
        hd[idx] = hd[idx - 1] | hd[idx];
    }
    if (lane >= 2)
    {
        if (!hd[idx] && ptr[idx - 2] < ptr[idx])
        {
            ptr[idx] = ptr[idx - 2];
            indexes[idx] = indexes[idx - 2];
        }
        hd[idx] = hd[idx - 2] | hd[idx];
    }
    if (lane >= 4)
    {
        if (!hd[idx] && ptr[idx - 4] < ptr[idx]) 
        {
            ptr[idx] = ptr[idx - 4];
            indexes[idx] = indexes[idx - 4];
        }
        hd[idx] = hd[idx - 4] | hd[idx];
    }
    if (lane >= 8)
    {
        if (!hd[idx] && ptr[idx - 8] < ptr[idx])
        {
            ptr[idx] = ptr[idx - 8];
            indexes[idx] = indexes[idx - 8];
        }
        hd[idx] = hd[idx - 8] | hd[idx];
    }
    if (lane >= 16)
    {
        if (!hd[idx] && ptr[idx - 16] < ptr[idx])
        {
            ptr[idx] = ptr[idx - 16];
            indexes[idx] = indexes[idx - 16];
        }
        hd[idx] = hd[idx - 16] | hd[idx];
    }
    return ptr[idx];
}

__device__ float segscan_min_block(volatile float* ptr, volatile int* hd, volatile int* indexes)
{
    const unsigned int idx = threadIdx.x;
    unsigned int warpid = idx >> 5;
    unsigned int warp_first = warpid << 5;
    unsigned int warp_last = warp_first + 31;
    // Step 1a:
    // Before overwriting the input head flags , record whether
    // this warp begins with an "open" segment.
    bool warp_is_open = (hd[warp_first] == 0);
    __syncthreads();
    // Step 1b:
    // Intra -warp segmented scan in each warp.
    float val = segscan_min_warp(ptr, hd, indexes);
    // Step 2a:
    // Since ptr [] contains *inclusive * results , irrespective of Kind ,
    // the last value is the correct partial result.
    float warp_total = ptr[warp_last];
    // Step 2b:
    // warp_flag is the OR -reduction of the flags in a warp and is
    // computed indirectly from the mindex values in hd [].
    // will_accumulate indicates that a thread will only accumulate a
    // partial result in Step 4 if there is no segment boundary to its left.
    bool warp_flag = hd[warp_last] != 0 || !warp_is_open;
    bool will_accumulate = warp_is_open && hd[idx] == 0;
    __syncthreads();
    // Step 2c: The last thread in each warp writes partial results
    if (idx == warp_last)
    {
        ptr[warpid] = warp_total;
        hd[warpid] = warp_flag;
        indexes[warpid] = indexes[warp_last];
    }
    __syncthreads();
    // Step 3: One warp scans the per -warp results
    if (warpid == 0)
        segscan_min_warp(ptr, hd, indexes);
    __syncthreads();
    // Step 4: Accumulate results from Step 3, as appropriate .
    if (warpid != 0 && will_accumulate && ptr[warpid - 1] < val)
    {
        val = ptr[warpid - 1];
        indexes[idx] = indexes[warpid - 1];
    }
    __syncthreads();
    ptr[idx] = val;
    __syncthreads();
    return val;
}

//__device__ void radixFragmentSort()


//__device__ void segmentScan(int* arr, int *shm_indexes)
//{
//    // up-sweep phase
//    int k = 2 * threadIdx.x, d;
//    for (d = 2; d < 2 * blockDim.x; d <<= 1)
//    {
//        
//        if (k & (~((d << 1) - 1))) // k % 2*d == 0
//        {
//            if (!IS_NEW_SEGMENT(shm_indexes[k + (d << 1) - 1]))
//                arr[k + (d << 1) - 1] += arr[k + d - 1];
//            scene.tree.flags[shm_indexes[k + (d << 1) - 1]] |= IS_NEW_SEGMENT(shm_indexes[k + d - 1]);
//        }
//        __syncthreads();
//    }
//    // down-sweep phase
//    if (threadIdx.x == blockDim.x - 1) arr[2 * blockDim.x - 1] = 0;
//    __syncthreads();
//    for (d >>= 1; d >= 0; d >>= 1)
//    {
//        if (k & (~((d << 1) - 1))) // k % 2*d == 0
//        {
//            int t = arr[k + d - 1];
//            arr[k + d - 1] = arr[k + (d << 1) - 1];
//            if (IS_NEW_SEGMENT(shm_indexes[k + d]))
//                arr[k + (d << 1) - 1] = 0;
//            else if (IS_NEW_SEGMENT(shm_indexes[k + d - 1]))
//                arr[k + (d << 1) - 1] = t;
//            else
//                arr[k + (d << 1) - 1] += t;
//            scene.tree.flags[shm_indexes[k + d - 1]] &= ~0b100;
//        }
//        __syncthreads();
//    }
//}
//
//__device__ void segmentScanByIntersectionCounts(int* arr, int *shm_indexes)
//{
//    // zamień N na 1, pozostałe 0
//    // up-sweep phase
//    int k = 2 * threadIdx.x, d;
//    if (arr[k] == GET_INTERSECTION_COUNT(shm_indexes[k])) arr[k] = 1;
//    else arr[k] = 0;
//    if (arr[k + blockDim.x] == GET_INTERSECTION_COUNT(shm_indexes[k + blockDim.x])) arr[k + blockDim.x] = 1;
//    else arr[k + blockDim.x] = 0;
//    for (d = 2; d < 2 * blockDim.x; d <<= 1)
//    {
//        if (k & (~((d << 1) - 1))) // k % 2*d == 0
//        {
//            if (!IS_NEW_SEGMENT(shm_indexes[k + (d << 1) - 1]))
//                arr[k + (d << 1) - 1] += arr[k + d - 1];
//            scene.tree.flags[shm_indexes[k + (d << 1) - 1]] |= IS_NEW_SEGMENT(shm_indexes[k + d - 1]);
//        }
//        __syncthreads();
//    }
//    // down-sweep phase
//    if (threadIdx.x == blockDim.x - 1) arr[2 * blockDim.x - 1] = 0;
//    __syncthreads();
//    for (d >>= 1; d >= 0; d >>= 1)
//    {
//        if (k & (~((d << 1) - 1))) // k % 2*d == 0
//        {
//            int t = arr[k + d - 1];
//            arr[k + d - 1] = arr[k + (d << 1) - 1];
//            if (IS_NEW_SEGMENT(shm_indexes[k + d]))
//                arr[k + (d << 1) - 1] = 0;
//            else if (IS_NEW_SEGMENT(shm_indexes[k + d - 1]))
//                arr[k + (d << 1) - 1] = t;
//            else
//                arr[k + (d << 1) - 1] += t;
//            scene.tree.flags[shm_indexes[k + d - 1]] &= ~0b100;
//        }
//        __syncthreads();
//    }
//}

__device__ void findIntersectionWithRay(int x, int y, int i, float* out_distances, int* out_indexes, float* shm_faces, int* shm_indexes, int* shm_intersectionIds, int* shm_a_to_scan)
{


    

    //if (shm_a_to_scan[idx + (idx / scene.tree.n)*scene.tree.n] == 1 && (idx % scene.tree.n == 0 || shm_a_to_scan[idx + (idx / scene.tree.n) * scene.tree.n - 1] == 0)) // uwaga na illegal access!!!!!!!
    //{
    //    // mamy wynik dla piksela (x,y)

    //    

    //    out_distances[x + y * scene.width] = shm_faces[idx + (idx / scene.tree.n) * scene.tree.n];
    //    out_indexes[x + y * scene.width] = shm_indexes[idx + (idx / scene.tree.n) * scene.tree.n] * (GET_OP_ID(scene.tree.flags[shm_indexes[idx + (idx / scene.tree.n) * scene.tree.n]]) - 1);

    //    //printf("Jest wynik: %d %d %f %d\n", x, y, shm_faces[idx + (idx / scene.tree.n) * scene.tree.n], out_indexes[x+y*scene.width]);
    //}
    //if (shm_a_to_scan[idx + (idx / scene.tree.n + 1) * scene.tree.n] == 1 && shm_a_to_scan[idx + (idx / scene.tree.n + 1) * scene.tree.n - 1] == 0)
    //{
    //    // mamy wynik dla piksela (x,y)

    //    

    //    out_distances[x + y * scene.width] = shm_faces[idx + (idx / scene.tree.n + 1) * scene.tree.n];
    //    out_indexes[x + y * scene.width] = shm_indexes[idx + (idx / scene.tree.n + 1) * scene.tree.n] * (GET_OP_ID(scene.tree.flags[shm_indexes[idx + (idx / scene.tree.n + 1) * scene.tree.n]]) - 1);

    //    //printf("Jest wynik: %d %d %f %d\n", x, y, shm_faces[idx + (idx / scene.tree.n + 1) * scene.tree.n], out_indexes[x+y*scene.width]);
    //}

    
    
    //unitNormal = make_float3(
    //    normalSign * (scene.observer.x - scene.tree.x[index] + vecTopointOnScene.x * distance) / scene.tree.radius[index],
    //    normalSign * (scene.observer.y - scene.tree.y[index] + vecTopointOnScene.y * distance) / scene.tree.radius[index],
    //    normalSign * (scene.observer.z - scene.tree.z[index] + vecTopointOnScene.z * distance) / scene.tree.radius[index]);
    //prodNL = scalarProduct(unitNormal, scene.unitVectorToLight);
    //unitReflected = normalised(make_float3(
    //    2 * prodNL * unitNormal.x - scene.unitVectorToLight.x,
    //    2 * prodNL * unitNormal.y - scene.unitVectorToLight.y,
    //    2 * prodNL * unitNormal.z - scene.unitVectorToLight.z));
    //prodVR = scalarProduct(unitReflected, unitToScene);

    //color = scene.tree.color[index];

    //return distance;
}

__global__ void calculateDistancesAndIndexes(float *out_distances, int* out_indexes)
{
    //extern __shared__ float data[];

    __shared__ float shm_faces[2*1024];
    __shared__ int shm_intersectionIds[2*1024];
    __shared__ int shm_indexes[2*1024];
    __shared__ int shm_a_to_scan[2*1024];
    //__shared__ int shm_perm[2 * 1024];
    
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int idx = threadIdx.x, idxPlusBlockDim = idx + blockDim.x;
    const int x = (index / scene.tree.n) % scene.width, y = (index / scene.tree.n) / scene.width, i = index % scene.tree.n;
    //if (threadIdx.x < scene.tree.n) // !!!!!!!!!
    //{
    //    data[threadIdx.x] = scene.tree.operationId[threadIdx.x];
    //    data[threadIdx.x + scene.tree.n] = scene.tree.operationId[threadIdx.x + scene.tree.n];
    //    data[threadIdx.x + 2 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 2*scene.tree.n];
    //    data[threadIdx.x + 3 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 3*scene.tree.n];
    //    data[threadIdx.x + 4 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 4*scene.tree.n];
    //    data[threadIdx.x + 5 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 5*scene.tree.n];
    //}
   /* scene.tree.operationId = data;
    scene.tree.color = scene.tree.operationId + scene.tree.n;
    scene.tree.x = (float*)(scene.tree.color + scene.tree.n);
    scene.tree.y = scene.tree.x + scene.tree.n;
    scene.tree.z = scene.tree.y + scene.tree.n;
    scene.tree.radius = scene.tree.z + scene.tree.n;*/

    if (y < scene.height)
    {
        //findIntersectionWithRay(x, y, i, out_distances, out_indexes, data, (int*)(data+2*blockDim.x), (int*)(data + 2 * blockDim.x) + 2 * blockDim.x, (int*)(data + 2 * blockDim.x) + 4 * blockDim.x);
        
        float3 vecTopointOnScene = make_float3(
            scene.planeDistance * scene.direction.x - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.x * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.y * scene.planeHorizontal.z - scene.direction.z * scene.planeHorizontal.y),
            scene.planeDistance * scene.direction.y - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.y * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.z * scene.planeHorizontal.x - scene.direction.x * scene.planeHorizontal.z),
            scene.planeDistance * scene.direction.z - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.z * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.x * scene.planeHorizontal.y - scene.direction.y * scene.planeHorizontal.x));
        //float3 unitToScene = minusNormalised(vecTopointOnScene);
        //float3 unitNormal, unitReflected;
        float distance = INFINITY;
        int index = 0, normalSign;

        //float faces[2];

        calculateIntersection(i, &shm_faces[idx], &shm_faces[idxPlusBlockDim], vecTopointOnScene);

        shm_intersectionIds[idx] = GET_INTERSECTION_ID(i);
        shm_intersectionIds[idxPlusBlockDim] = GET_INTERSECTION_ID(i);

        shm_indexes[idx] = i;// | (scene.tree.flags[i] & 0xffff0000);
        shm_indexes[idxPlusBlockDim] = i;// | (scene.tree.flags[i] & 0xffff0000);

        /*shm_perm[idx] = idx;
        shm_perm[idxPlusBlockDim] = idxPlusBlockDim;*/

        if (shm_faces[idx] == INFINITY)
        {
            shm_a_to_scan[idx] = 0;
            shm_a_to_scan[idxPlusBlockDim] = 0;
        }
        else if (GET_OP_ID(scene.tree.flags[i]) == OP_INTERSECTION)
        {
            shm_a_to_scan[idx] = 1;
            shm_a_to_scan[idxPlusBlockDim] = -1;
        }
        else
        {
            shm_a_to_scan[idx] = -1;
            shm_a_to_scan[idxPlusBlockDim] = 1;
        }

        // dodać shm_a_to_scan to poniższych funkcji sortujących!
        fragmentSortInBlock(shm_faces, shm_indexes, shm_intersectionIds, shm_a_to_scan);
        fragmentSortInBlock(shm_faces + blockDim.x, shm_indexes + blockDim.x, shm_intersectionIds + blockDim.x, shm_a_to_scan + blockDim.x);



        mergeTwoBlockSizeArrays(shm_faces, shm_indexes, shm_intersectionIds, shm_a_to_scan);


        /*  __syncthreads();
         if (x == 300 && y == 300 && i == 0)
         {
             for (int j = 0; j < 2 * blockDim.x; j++)
                 printf("%f;%d;%d;%d\n", shm_faces[j], shm_indexes[j], GET_INTERSECTION_COUNT(shm_indexes[idx]), shm_intersectionIds[j]);
         }
         __syncthreads();*/

        int setIdx = 0, setIdxPlusBlockDim = 0;
        if (idx == 0 || shm_intersectionIds[idx] > shm_intersectionIds[idx - 1])
        {
            setIdx = 1;
        }

        if (shm_intersectionIds[idxPlusBlockDim] > shm_intersectionIds[idxPlusBlockDim - 1])
        {
            setIdxPlusBlockDim = 1;
        }

        shm_intersectionIds[idx] = setIdx;
        shm_intersectionIds[idxPlusBlockDim] = setIdxPlusBlockDim;


        // wybierz najmniejsze takie i, że scan = liczba przecięć
        segscan_block(shm_a_to_scan, shm_intersectionIds);
        segscan_block(shm_a_to_scan + blockDim.x, shm_intersectionIds + blockDim.x);




        shm_intersectionIds[idx] = setIdx;
        shm_intersectionIds[idxPlusBlockDim] = setIdxPlusBlockDim;
        /*if (x == 0 && y == 0 && i == 0)
        {
            for (int j = 0; j < 2 * blockDim.x; j++)
            {
                printf("(%f,%d,%d),\t", shm_faces[j], shm_indexes[j], shm_intersectionIds[j]);
            }
            printf("\n");
        }*/


        //if (shm_a_to_scan[idx] == GET_INTERSECTION_COUNT(shm_indexes[idx])) shm_a_to_scan[idx] = 1;
        //else shm_a_to_scan[idx] = 0;

        //if (shm_a_to_scan[idxPlusBlockDim] == GET_INTERSECTION_COUNT(shm_indexes[idxPlusBlockDim])) shm_a_to_scan[idxPlusBlockDim] = 1;
        //else shm_a_to_scan[idxPlusBlockDim] = 0;



        //segscan_block(shm_a_to_scan, shm_intersectionIds);
        //segscan_block(shm_a_to_scan + blockDim.x, shm_intersectionIds + blockDim.x);




        //if (!(shm_a_to_scan[idx] == 1 && (setIdx || shm_a_to_scan[idx - 1] == 0)))
        //{
        //    shm_faces[idx] = INFINITY;
        //}
        //if (!(shm_a_to_scan[idxPlusBlockDim] == 1 && (setIdxPlusBlockDim || shm_a_to_scan[idxPlusBlockDim - 1] == 0)))
        //{
        //    shm_faces[idxPlusBlockDim] = INFINITY;
        //}
        //shm_intersectionIds[idx] = idx % (2 * scene.tree.n) == 0 ? 1 : 0;
        //shm_intersectionIds[idxPlusBlockDim] = (idxPlusBlockDim) % (2 * scene.tree.n) == 0 ? 1 : 0;


        if (shm_a_to_scan[idx] != GET_INTERSECTION_COUNT(shm_indexes[idx])) shm_faces[idx] = INFINITY;
        //else shm_a_to_scan[idx] = 0;

        if (shm_a_to_scan[idxPlusBlockDim] != GET_INTERSECTION_COUNT(shm_indexes[idxPlusBlockDim])) shm_faces[idxPlusBlockDim] = INFINITY;
        //else shm_a_to_scan[idxPlusBlockDim] = 0;

        shm_intersectionIds[idx] = idx % (2 * scene.tree.n) == 0 ? 1 : 0;
        shm_intersectionIds[idxPlusBlockDim] = (idxPlusBlockDim) % (2 * scene.tree.n) == 0 ? 1 : 0;

        segscan_min_block(shm_faces, shm_intersectionIds, shm_indexes);
        segscan_min_block(shm_faces + blockDim.x, shm_intersectionIds + blockDim.x, shm_indexes + blockDim.x);

        /* __syncthreads();
        if (x == 300 && y == 300 && i == 0)
        {
            for (int j = 0; j < 2 * blockDim.x; j++)
                printf("%f;%d;%d;%d\n", shm_faces[j], shm_indexes[j], GET_INTERSECTION_COUNT(shm_indexes[idx]), shm_intersectionIds[j]);
        }
        __syncthreads();*/

        out_distances[x + y * scene.width] = INFINITY;
        if (idx % scene.tree.n == scene.tree.n - 1)
        {
            out_distances[x + y * scene.width] = shm_faces[2 * idx + 1];
            out_indexes[x + y * scene.width] = shm_indexes[2 * idx + 1];
            //if (y == 0) printf("%d;%f\n", x, shm_faces[2 * idx + 1]);
        }
    }
}

__global__ void renderView(float* in_distances, int* in_indexes_out_pixels)
{
    //extern __shared__ float data[];
   // __shared__ float data[1024 * (2 + 6)];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    //if (threadIdx.x < scene.tree.n) // !!!!!!!!!
    //{
    //    data[threadIdx.x] = scene.tree.operationId[threadIdx.x];
    //    data[threadIdx.x + scene.tree.n] = scene.tree.operationId[threadIdx.x + scene.tree.n];
    //    data[threadIdx.x + 2 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 2*scene.tree.n];
    //    data[threadIdx.x + 3 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 3*scene.tree.n];
    //    data[threadIdx.x + 4 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 4*scene.tree.n];
    //    data[threadIdx.x + 5 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 5*scene.tree.n];
    //}
   /* scene.tree.operationId = data;
    scene.tree.color = scene.tree.operationId + scene.tree.n;
    scene.tree.x = (float*)(scene.tree.color + scene.tree.n);
    scene.tree.y = scene.tree.x + scene.tree.n;
    scene.tree.z = scene.tree.y + scene.tree.n;
    scene.tree.radius = scene.tree.z + scene.tree.n;*/

    int x = index % scene.width, y = index / scene.width;
    if (y < scene.height)
    {
        int normalSign = 1;
        if (in_indexes_out_pixels[index] < 0)
        {
            normalSign = -1;
            in_indexes_out_pixels[index] = -in_indexes_out_pixels[index];
        }
        float distance = in_distances[index], radius = scene.tree.radius[in_indexes_out_pixels[index]];
        float3 vecTopointOnScene = make_float3(
            scene.planeDistance * scene.direction.x - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.x * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.y * scene.planeHorizontal.z - scene.direction.z * scene.planeHorizontal.y),
            scene.planeDistance * scene.direction.y - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.y * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.z * scene.planeHorizontal.x - scene.direction.x * scene.planeHorizontal.z),
            scene.planeDistance * scene.direction.z - (1.0f - 2.0f * x / scene.width) * scene.planeHorizontal.z * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * y / scene.height) * scene.planeHeight / 2 * (scene.direction.x * scene.planeHorizontal.y - scene.direction.y * scene.planeHorizontal.x));
        float3 unitToScene = minusNormalised(vecTopointOnScene);
        float3 unitNormal = make_float3(
            normalSign * (scene.observer.x - scene.tree.x[in_indexes_out_pixels[index]] + vecTopointOnScene.x * distance) / radius,
            normalSign * (scene.observer.y - scene.tree.y[in_indexes_out_pixels[index]] + vecTopointOnScene.y * distance) / radius,
            normalSign * (scene.observer.z - scene.tree.z[in_indexes_out_pixels[index]] + vecTopointOnScene.z * distance) / radius);
        float prodNL = scalarProduct(unitNormal, scene.unitVectorToLight);
        float3 unitReflected = normalised(make_float3(
            2 * prodNL * unitNormal.x - scene.unitVectorToLight.x,
            2 * prodNL * unitNormal.y - scene.unitVectorToLight.y,
            2 * prodNL * unitNormal.z - scene.unitVectorToLight.z));
        float prodVR = scalarProduct(unitReflected, unitToScene);

        int color = scene.tree.color[in_indexes_out_pixels[index]];

        if (prodNL < 0) prodNL = 0;
        if (prodVR < 0) prodVR = 0;
        prodVR = powf(prodVR, scene.light.m);
        float r = ((color & 0xff0000) / (float)0xff0000) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR,
            g = ((color & 0xff00) / (float)0xff00) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR,
            b = ((color & 0xff) / (float)0xff) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR;

        if (distance < INFINITY)
        {
            //printf("Koloruje %d, %d, %f, %f\n", x, y, prodNL, prodVR);
            in_indexes_out_pixels[index] = 0xff000000 | ((r > 1.0f ? 0xff : (int)(r * 0xff)) << 16) | ((g > 1.0f ? 0xff : (int)(g * 0xff)) << 8) | (b > 1.0f ? 0xff : (int)(b * 0xff));
        }
        else in_indexes_out_pixels[index] = 0xff000000;

    }
}

//__global__ void calculateDistancesAndIndexes(int* distances, int* indexes)
//{
//    extern __shared__ float data[];
//    int index = threadIdx.x + blockIdx.x * blockDim.x;
//
//    //if (threadIdx.x < scene.tree.n) // !!!!!!!!!
//    //{
//    //    data[threadIdx.x] = scene.tree.operationId[threadIdx.x];
//    //    data[threadIdx.x + scene.tree.n] = scene.tree.operationId[threadIdx.x + scene.tree.n];
//    //    data[threadIdx.x + 2 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 2*scene.tree.n];
//    //    data[threadIdx.x + 3 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 3*scene.tree.n];
//    //    data[threadIdx.x + 4 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 4*scene.tree.n];
//    //    data[threadIdx.x + 5 * scene.tree.n] = scene.tree.operationId[threadIdx.x + 5*scene.tree.n];
//    //}
//   /* scene.tree.operationId = data;
//    scene.tree.color = scene.tree.operationId + scene.tree.n;
//    scene.tree.x = (float*)(scene.tree.color + scene.tree.n);
//    scene.tree.y = scene.tree.x + scene.tree.n;
//    scene.tree.z = scene.tree.y + scene.tree.n;
//    scene.tree.radius = scene.tree.z + scene.tree.n;*/
//
//    int x = (index / scene.tree.n) % scene.width, y = (index / scene.tree.n) / scene.width, i = index % scene.tree.n;
//    if (y < scene.height)
//    {
//        float prodNL, prodVR;
//        int color;
//        float distance = findIntersectionWithRay(x, y, i, prodNL, prodVR, color, data, (int*)(data + 2 * blockDim.x), (int*)(data + 2 * blockDim.x) + 2 * blockDim.x, (int*)(data + 2 * blockDim.x) + 4 * blockDim.x);
//        if (prodNL < 0) prodNL = 0;
//        if (prodVR < 0) prodVR = 0;
//        prodVR = powf(prodVR, scene.light.m);
//        float r = ((color & 0xff0000) / (float)0xff0000) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR,
//            g = ((color & 0xff00) / (float)0xff00) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR,
//            b = ((color & 0xff) / (float)0xff) * (scene.light.ka + scene.light.kd * prodNL) + scene.light.ks * prodVR;
//
//        if (distance < INFINITY)
//        {
//            printf("Koloruje %d, %d, %f, %f\n", x, y, prodNL, prodVR);
//            out[index] = 0xff000000 | ((r > 1.0f ? 0xff : (int)(r * 0xff)) << 16) | ((g > 1.0f ? 0xff : (int)(g * 0xff)) << 8) | (b > 1.0f ? 0xff : (int)(b * 0xff));
//        }
//        else out[index] = 0xff000000;
//    }
//}

void allocAndCreateTree(normalisedCsgTree& tree, int **pd_data, int *flag_data, int *color_data, float *shape_data, int n)
{
    tree.n = n;
    hipError_t err = hipMalloc(pd_data, TREE_SIZE(tree));
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
    }
    hipMemcpy(*pd_data, flag_data, sizeof(int) * tree.n, hipMemcpyHostToDevice);
    hipMemcpy(*pd_data + tree.n, color_data, sizeof(int) * tree.n, hipMemcpyHostToDevice);
    hipMemcpy(*pd_data + 2 * tree.n, shape_data, 4 * sizeof(float) * tree.n, hipMemcpyHostToDevice);
    tree.flags = *pd_data;
    tree.color = tree.flags + tree.n;
    tree.x = (float*)(tree.color + tree.n);
    tree.y = tree.x + tree.n;
    tree.z = tree.y + tree.n;
    tree.radius = tree.z + tree.n;
}

extern "C" void __declspec(dllexport) __stdcall GPURender
(
    int h_out[],
    int width,
    int height,
    float cameraParams[9],
    float planeDistance,
    float planeHeight,
    float lightParams[6],
    //int tree_shapeIds[],
    int tree_flags[],
    int color_data[],
    float raw_spheres[],
    int treeSize
)
{
    struct sceneParameters sc{};
    sc.observer = make_float3(cameraParams[0], cameraParams[1], cameraParams[2]);
    sc.direction = make_float3(cameraParams[3], cameraParams[4], cameraParams[5]);
    sc.planeHorizontal = make_float3(cameraParams[6], cameraParams[7], cameraParams[8]);
    sc.width = width;
    sc.height = height;
    sc.planeDistance = planeDistance;
    sc.planeHeight = planeHeight;
    sc.unitVectorToLight = make_float3(cos(lightParams[1]) * cos(lightParams[0]), cos(lightParams[1]) * sin(lightParams[0]), -sin(lightParams[1]));
    sc.light.ka = lightParams[2];
    sc.light.kd = lightParams[3];
    sc.light.ks = lightParams[4];
    sc.light.m = lightParams[5];
    
    int* d_out, *d_tdata;
    float* d_dist;
    allocAndCreateTree(sc.tree, &d_tdata, tree_flags, color_data, raw_spheres, treeSize);
    hipMalloc(&d_out, width * height * sizeof(int));
    hipMalloc(&d_dist, width* height * sizeof(float));
    hipMemset(d_out, 0, width * height * sizeof(int)); 
    hipMemset(d_dist, 0xff, width* height * sizeof(float));
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(scene), &sc, sizeof(struct sceneParameters));
    int blockDim = (1024 / sc.tree.n) * sc.tree.n;
    int blocks = ceil((float)width * height * sc.tree.n / blockDim);
    //int shmSize = (2 * sizeof(float) + 6 * sizeof(int)) * blockDim; //TREE_SIZE(sc.tree);
    calculateDistancesAndIndexes<<<blocks, blockDim/*, shmSize*/>>>(d_dist, d_out);
    blockDim = 1024;
    blocks = ceil((float)width * height / blockDim);
    renderView<<<blocks, blockDim>>>(d_dist, d_out);
    hipMemcpy(h_out, d_out, width * height * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_tdata);
    hipFree(d_dist);
}

//__global__ void calculateIntersection2(int offset, float* faces, float* backFace)
//{
//    const int index = offset + threadIdx.x + blockDim.x * blockIdx.x;
//    int xIm = (index / scene.tree.n) % scene.width, yIm = (index / scene.tree.n) / scene.width, i = index % scene.tree.n;
//    float x = scene.tree.x[i], y = scene.tree.y[i], z = scene.tree.z[i], radius = scene.tree.radius[i];
//
//    float3 vecTopointOnScene = make_float3(
//        scene.planeDistance * scene.direction.x - (1.0f - 2.0f * xIm / scene.width) * scene.planeHorizontal.x * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * yIm / scene.height) * scene.planeHeight / 2 * (scene.direction.y * scene.planeHorizontal.z - scene.direction.z * scene.planeHorizontal.y),
//        scene.planeDistance * scene.direction.y - (1.0f - 2.0f * xIm / scene.width) * scene.planeHorizontal.y * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * yIm / scene.height) * scene.planeHeight / 2 * (scene.direction.z * scene.planeHorizontal.x - scene.direction.x * scene.planeHorizontal.z),
//        scene.planeDistance * scene.direction.z - (1.0f - 2.0f * xIm / scene.width) * scene.planeHorizontal.z * scene.planeHeight / 2 * scene.width / scene.height + (1.0f - 2.0f * yIm / scene.height) * scene.planeHeight / 2 * (scene.direction.x * scene.planeHorizontal.y - scene.direction.y * scene.planeHorizontal.x));
//
//    //http://paulbourke.net/geometry/circlesphere/index.html#linesphere
//    float a = vecTopointOnScene.x * vecTopointOnScene.x + vecTopointOnScene.y * vecTopointOnScene.y + vecTopointOnScene.z * vecTopointOnScene.z,
//        b = 2 * (vecTopointOnScene.x * (scene.observer.x - x) + vecTopointOnScene.y * (scene.observer.y - y) + vecTopointOnScene.z * (scene.observer.z - z)),
//        c = x * x + y * y + z * z + scene.observer.x * scene.observer.x + scene.observer.y * scene.observer.y + scene.observer.z * scene.observer.z - 2 * (x * scene.observer.x + y * scene.observer.y + z * scene.observer.z) - radius * radius;
//
//    float delta = b * b - 4 * a * c;
//    if (delta >= 0)
//    {
//        faces[2*index] = (-b - sqrt(delta)) / 2 / a;
//        faces[2 * index+1] = (-b + sqrt(delta)) / 2 / a;
//    }
//    else
//    {
//        faces[2 * index] = INFINITY;
//        faces[2 * index+1] = -INFINITY;
//    }
//}
//
//extern "C" void __declspec(dllexport) __stdcall GPUThrustRender
//(
//    int h_out[],
//    int width,
//    int height,
//    float cameraParams[9],
//    float planeDistance,
//    float planeHeight,
//    float lightParams[6],
//    //int tree_shapeIds[],
//    int tree_flags[],
//    int color_data[],
//    float raw_spheres[],
//    int treeSize
//)
//{
//    const long long maxMemory = 1 << 30;
//
//    struct sceneParameters sc {};
//    sc.observer = make_float3(cameraParams[0], cameraParams[1], cameraParams[2]);
//    sc.direction = make_float3(cameraParams[3], cameraParams[4], cameraParams[5]);
//    sc.planeHorizontal = make_float3(cameraParams[6], cameraParams[7], cameraParams[8]);
//    sc.width = width;
//    sc.height = height;
//    sc.planeDistance = planeDistance;
//    sc.planeHeight = planeHeight;
//    sc.unitVectorToLight = make_float3(cos(lightParams[1]) * cos(lightParams[0]), cos(lightParams[1]) * sin(lightParams[0]), -sin(lightParams[1]));
//    sc.light.ka = lightParams[2];
//    sc.light.kd = lightParams[3];
//    sc.light.ks = lightParams[4];
//    sc.light.m = lightParams[5];
//
//    long long requiredMemory = width * height * sc.tree.n;
//
//    int* d_out, * d_tdata;
//    float* d_dist;
//    allocAndCreateTree(sc.tree, &d_tdata, tree_flags, color_data, raw_spheres, treeSize);
//    hipMalloc(&d_out, width * height * sizeof(int));
//    hipMalloc(&d_dist, width * height * sizeof(float));
//    hipMemset(d_out, 0, width * height * sizeof(int));
//    hipMemset(d_dist, 0xff, width * height * sizeof(float));
//    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(scene), &sc, sizeof(struct sceneParameters));
//    int blockDim = (1024 / sc.tree.n) * sc.tree.n;
//    int blocks = ceil((float)width * height * sc.tree.n / blockDim);
//    int shmSize = (2 * sizeof(float) + 6 * sizeof(int)) * blockDim; //TREE_SIZE(sc.tree);
//    calculateDistancesAndIndexes << <blocks, blockDim, shmSize >> > (d_dist, d_out);
//    blockDim = 1024;
//    blocks = ceil((float)width * height / blockDim);
//    renderView << <blocks, blockDim >> > (d_dist, d_out);
//    hipMemcpy(h_out, d_out, width * height * sizeof(int), hipMemcpyDeviceToHost);
//    hipFree(d_out);
//    hipFree(d_tdata);
//    hipFree(d_dist);
//}


int main()
{
    int width = 1000, height = 1000;
    /*int tree_flags[]{ MAKE_FLAG(0,2,1,1), MAKE_FLAG(0,2,0,1) };
    float raw_spheres[]{ 0, 0.5f, 0, 0, 1.5f, 1.5f, 0.5f, 0.5f };*/
    int tree_flags[]{ 5,1,65541,65537 };
    float raw_spheres[]{ 0.5,0.25,0,0.25,0,0.3,0,0.3,1.5,1.5,1.5,1.5,0.5,0.5,0.5,0.5 };
    float cameraParams[]{ 0,0,0,0,0,1,0,1,0 };
    int color_data[]{ 0xffffffff,0xffff0000, 0xffffffff, 0xffffffff };
    float lightParams[]{ 0,0,0.1,0.45,0.45,30 };
    int* h_out;
    hipHostAlloc(&h_out, width * height * sizeof(int), hipHostMallocDefault);
    GPURender(h_out, width, height, cameraParams, 0.5, 0.5, lightParams, tree_flags, color_data, raw_spheres, 4);
    hipHostFree(h_out);
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) printf("CUDA error: %s", hipGetErrorString(err)); // add
    return 0;
}

